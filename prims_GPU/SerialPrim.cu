
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <cstring> 
using namespace std;



// R1 source, R2 destination,R3 weight
void initMSTEdgeList(int *R1 , int *R2 , int *R3, int *indexArray , int *destinationVertexArray, int *weightArray, int vertices)
{
	for(int v = indexArray[0] ; v < indexArray[1] ; v++)
	{
		// update weight of adjacent vertices of vertex 0. -1 bcz counting at r2 starts from 1 i.e at index 0 vertex 1 is placed.
		// So correspondingly for R3.
		R3[destinationVertexArray[v]-1] = weightArray[v];
	}	

	for(int i = 0;i< vertices-1;i++)
	{
		R2[i] = i+1;
		R1[i] = 0;
	}
}

void initializeGraph(int *indexArray , int *destinationVertexArray , int *weightArray, int edges)
{

	
	int u = 1,v,w;
	int count = 0;
	int prev = 1;
    for(int i = 0 ; i < edges ; i++) 
	{
	   prev = u;
       	   cin >>u>>v>>w;
	   if(prev == u)
	   {
			count++;
	   }	
	   else
	   {
		   indexArray[prev] = count + indexArray[prev-1];
		   count = 1;
	   }	
	   destinationVertexArray[i] = v-1;
	   weightArray[i] = w;
    }

}


void printArray(char name,int *a  , int size)
{
    printf("%c: ",name);
    for(int i = 0; i < size ; i++)
        printf("%d\t",a[i]);
    printf("\n");
}


void updateMSTList(int startIndex,int endIndex, int *nearestVertex, int *destinationVertexArray,int *weightArray,int *R1,int *R2,int *R3)
{
	
	for(int i = startIndex; i <= endIndex;i++)
	{	
		if(destinationVertexArray[i] != 0)
		{	
			if(R2[destinationVertexArray[i]-1] != 0)
			{	// -1 bcz the r2 & r3 counting begins from 1.// update the edge and the source vertex.
				if(R3[destinationVertexArray[i]-1] > weightArray[i] )
				{		
					R3[destinationVertexArray[i]-1] = weightArray[i];
					R1[destinationVertexArray[i]-1] = *nearestVertex;
				}
			}
		}
	}
}

void findNearestVertex(int *R1, int *R2 , int *R3, int *nearest_vertex,int vertices)
{

	int *min = (int *)malloc(sizeof(int));
	memset(min,1,sizeof(int));	

	for(int i = 0; i < vertices-1 ; i++)
	{
		if(R3[i] < (*min) && R2[i] != 0)
		{
			 *min = R3[i];
			*nearest_vertex = i+1;			
		}		
	}
	R2[(*nearest_vertex)-1] = 0;
}



int main()
{
	// graph representatioon arrays
	int *indexArray;
	int *weightArray;
	int *destinationVertexArray;


	// MST Edge list
	int *R1;
	int *R2;
	int *R3;
	int *nearestVertex;
   	

	
	int edges,vertices;
	cin>>vertices>>edges;
	edges = edges/2;	
	
	// iteration indexes for updation
	int startIndex, endIndex;
	
	// allocate memory to compact adjacency list

	indexArray = (int *)malloc(vertices * sizeof(int));
 	weightArray = (int *)malloc(2 * edges * sizeof(int));
    destinationVertexArray = (int *)malloc(2 * edges * sizeof(int));
	nearestVertex = (int *)malloc(sizeof(int));
	
	memset(indexArray,0,vertices * sizeof(int));
	memset(weightArray,0,2 * edges * sizeof(int));
	memset(destinationVertexArray,0,2 * edges * sizeof(int));
	*nearestVertex = 0;
	
	R1 = (int *)malloc((vertices-1) * sizeof(int));
	R2 = (int *)malloc((vertices-1) * sizeof(int));
	R3 = (int *)malloc((vertices-1) * sizeof(int));
	memset(R3,1,(vertices-1)*sizeof(int));
	
	initializeGraph(indexArray,destinationVertexArray,weightArray,2*edges);
	
	initMSTEdgeList(R1,R2,R3,indexArray,destinationVertexArray,weightArray,vertices);	


	
		hipEvent_t start, stop;
        float time;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord( start, 0 );
	
	// kernel launch
	for(int v = 0 ; v < vertices-2; v++)
	{
		findNearestVertex(R1, R2, R3, nearestVertex,vertices);

		startIndex = indexArray[*nearestVertex];
		if(*(nearestVertex) == vertices-1)
        		endIndex = 2*edges-1;
	    	else
        		endIndex = indexArray[*nearestVertex+1]-1;

		updateMSTList(startIndex, endIndex , nearestVertex, destinationVertexArray, weightArray, R1, R2, R3);
		
	}

        printf("u\tv\tw\n");
		long cost = 0;
		hipEventRecord( stop, 0 );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &time, start, stop );
        hipEventDestroy( start );
        hipEventDestroy( stop );

        for(int i = 0; i < vertices-1;i++)
        {
       		 printf("%d\t%d\t%d\t\n",R1[i]+1,i+2,R3[i]);
		 cost = cost + R3[i];
        }
		cout<<"\ncost:"<<cost;
 		printf("\nTime: %fms\n",time);		
}
   

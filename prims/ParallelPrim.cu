
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include <fstream>
#include <iostream>

using namespace std;

#define THREADS_PER_BLOCK 1024
#define NUMBER_OF_BLOCKS 1024

#define DEBUG 0

//initially, 6 & 9

// number of threads_per_block*blocks should be atleast V-1.
// threads per block should be greater than or equal to number_of_blocks and both should be power of 2.
// only bcz of the restriction of cuda kernel two.


// if the vertices begin with 1 not 0, be careful, got to map it accordingly.
//destinationVertexArray decrease the value of indices by 1, and when result is out, increase the indices by 1.
//R2 array will also begin with 2 in that case, as the R1 initialization would be 1.

void initializeGraph(int *indexArray , int *destinationVertexArray , int *weightArray, int edges, int vertices)
{

	// to be read from file or generators for now make static.

	
	int u = 1,v,w;
	int count = 0;
	int prev = 1;
	for(int i = 0 ; i < edges ; i++) 
	{
		prev = u;
       	cin >>u>>v>>w;
	   	if(prev == u)
	   	{
			count++;
		}	
	   	else
	   	{
		   indexArray[prev] = count + indexArray[prev-1];
		   count = 1;
	   	}	
	   	destinationVertexArray[i] = v-1;
	   	weightArray[i] = w;
    }

}




// R1 source, R2 destination,R3 weight
void initMSTEdgeList(int *R1 , int *R2 , int *R3, int *indexArray , int *destinationVertexArray, int *weightArray, int vertices)
{
	for(int v = indexArray[0] ; v < indexArray[1] ; v++)
	{
		// update weight of adjacent vertices of vertex 0. -1 bcz counting at r2 starts from 1 i.e at index 0 vertex 1 is placed.
		// So correspondingly for R3.
		R3[destinationVertexArray[v]-1] = weightArray[v];
	}	

	for(int i = 0;i< vertices-1;i++)
	{
		R2[i] = i+1;
		R1[i] = 0;
	}
}


void printArray(int *a  , int size)
{
    for(int i = 0; i < size ; i++)
        printf("%d\t",a[i]);
    printf("\n");
}


__global__ void updateMSTList(int startIndex,int endIndex, int *d_nearestVertex, int *destinationVertexArray,int *weightArray,int *d_R1,int *d_R2,int *d_R3)
{
	int threadId = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
	int offset = startIndex;
	
	if((threadId+offset) >= startIndex && (threadId+offset) <= endIndex)
	{
		int destVertex = destinationVertexArray[threadId+offset];
		if(destVertex != 0)
		{
			if(d_R2[destVertex-1] != 0)
			{
				if(d_R3[destVertex-1] > weightArray[threadId+offset] )
				{
					d_R3[destVertex-1] = weightArray[threadId+offset];
					d_R1[destVertex-1] = *d_nearestVertex;
				}
			}
		}
	}
}

__global__ void findNearestVertexKernelOne( int *d_R2 , int *d_R3, int *T1, int *T2, int *d_nearestVertex,int secondKernelRequired,int vertices)
{
	int blockNo = blockIdx.x;
	int threadId =  blockNo * THREADS_PER_BLOCK + threadIdx.x;

	__shared__ int	r2[THREADS_PER_BLOCK],r3[THREADS_PER_BLOCK];	
  
	r2[threadIdx.x] = 0;	
	r3[threadIdx.x] = 10000;	

	if(threadId < vertices-1)
	{
		r2[threadIdx.x] = d_R2[threadId];
		r3[threadIdx.x] = d_R3[threadId];
	
	__syncthreads();
	
	
	int i = 0;
	for(i = THREADS_PER_BLOCK/2; i > 0 ; i = i>>1)
	{
		if(threadIdx.x < i)
		{
			if(r2[threadIdx.x] != 0 && r2[i + threadIdx.x]!=0)
			{
				// we also want to store the indices of the minimum location, so that we can get the desired vertex.
				if(r3[threadIdx.x] > r3[i+threadIdx.x])
				{
					//update index.
					r2[threadIdx.x] = r2[i+threadIdx.x];
					//update weight.
					r3[threadIdx.x] = r3[i+threadIdx.x];
				}	
				//else its the same.
			}
			else if(r2[threadIdx.x] == 0 && r2[i+threadIdx.x] == 0)
			{
				// do nothing
			}
			else if(r2[threadIdx.x] == 0 && r2[i+threadIdx.x] != 0)
			{
				r2[threadIdx.x] = r2[i+threadIdx.x];
				r3[threadIdx.x] = r3[i+threadIdx.x];
			}
			// else same present value needs to be considered in the 4th case.
		}
		__syncthreads();
				
	}	
		// only one value needed, first thread will write.
		if(threadIdx.x == 0)
		{
			if(!secondKernelRequired)
			{
				// update the nearest_Vertex
				*d_nearestVertex = r2[0];
			    // mark vertex as used..
			 	 d_R2[r2[0]-1] = 0;
			}
			// for multiple blocks, because they are in the shared memory, set T1 T2 here which is index and weight for second kernel..		
			else
			{
				T1[blockIdx.x] = r2[0];
				T2[blockIdx.x] = r3[0];
			}	
		}
			
	}


}


__global__ void findNearestVertexKernelTwo(int *d_R1 , int *d_R2 , int *d_R3 , int *d_T1, int *d_T2, int *d_nearestVertex)
{

	// no need to use the shared memory, only one block.	
	
    for(int i = NUMBER_OF_BLOCKS/2; i > 0 ; i = i>>1)
    {
        if(threadIdx.x < i)
        {
            if(d_T1[threadIdx.x] != 0 && d_T1[i + threadIdx.x]!=0)
            {
                // we also want to store the indices of the minimum location, so that we can get the desired vertex.
                if(d_T2[threadIdx.x] > d_T2[i+threadIdx.x])
                {
                    //update index.
                    d_T1[threadIdx.x] = d_T1[i+threadIdx.x];
                    //update weight.
                    d_T2[threadIdx.x] = d_T2[i+threadIdx.x];
                }
                //else its the same.
            }
            else if(d_T1[threadIdx.x] == 0 && d_T1[i+threadIdx.x] == 0)
            {
            	// do nothing
			}
            else if(d_T1[threadIdx.x] == 0 && d_T1[i+threadIdx.x] != 0)
            {
                d_T1[threadIdx.x] = d_T1[i+threadIdx.x];
                d_T2[threadIdx.x] = d_T2[i+threadIdx.x];
            }
            // else same present value needs to be considered in the 4th case.
        }
        __syncthreads();

    }

	if(threadIdx.x == 0)
    {
		*d_nearestVertex = d_T1[0];
         d_R2[d_T1[0]-1] = 0;
    }

}


int main()
{
	// graph representatioon arrays
	int *h_indexArray;
	int *h_weightArray;
	int *h_destinationVertexArray;

	int secondKernelRequired = 0;

	// device Graph	
	int *d_indexArray;
	int *d_weightArray;
	int *d_destinationVertexArray;


	// MST Edge list
	int *h_R1;
	int *h_R2;
	int *h_R3;
	int *h_nearestVertex;
   	int *h_T1;
	int *h_T2;
 
	// device MST list	
	int *d_R1;
	int *d_R2;
	int *d_R3;
	int *d_nearestVertex;
   	int *d_T1;
	int *d_T2;


	int edges,vertices;
    cin>>vertices>>edges;	

	cout << vertices << edges<< endl;
	edges = edges/2;		
	// iteration indexes for updation
	int startIndex, endIndex;
	
	// allocate memory to compact adjacency list

	h_indexArray = (int *)malloc(vertices * sizeof(int));
    h_weightArray = (int *)malloc(2 * edges * sizeof(int));
    h_destinationVertexArray = (int *)malloc(2 * edges * sizeof(int));
	h_nearestVertex = (int *)malloc(sizeof(int));
	
	memset(h_indexArray,0,vertices * sizeof(int));
	memset(h_weightArray,0,2 * edges * sizeof(int));
	memset(h_destinationVertexArray,0,2 * edges * sizeof(int));
	*h_nearestVertex = 0;
	
	h_R1 = (int *)malloc((vertices-1) * sizeof(int));
	h_R2 = (int *)malloc((vertices-1) * sizeof(int));
	h_R3 = (int *)malloc((vertices-1) * sizeof(int));

	h_T1 = (int *)malloc(NUMBER_OF_BLOCKS * sizeof(int));
	h_T2 = (int *)malloc(NUMBER_OF_BLOCKS * sizeof(int));
	
	memset(h_R3,1,(vertices-1)*sizeof(int));
	memset(h_T1,0,NUMBER_OF_BLOCKS * sizeof(int));
	memset(h_T2,0,NUMBER_OF_BLOCKS * sizeof(int));

	initializeGraph(h_indexArray,h_destinationVertexArray,h_weightArray,2*edges,vertices);


	
	initMSTEdgeList(h_R1,h_R2,h_R3,h_indexArray,h_destinationVertexArray,h_weightArray,vertices);	


	// cuda memory allocation of graph representation
	hipMalloc(&d_indexArray,(vertices) * sizeof(int));
	hipMalloc(&d_weightArray,2 * edges * sizeof(int));
	hipMalloc(&d_destinationVertexArray,2 * edges * sizeof(int));
	hipMalloc(&d_nearestVertex,sizeof(int));

	// cuda memory allocation of MST list
	hipMalloc(&d_R1,(vertices-1) * sizeof(int));
	hipMalloc(&d_R2,(vertices-1) * sizeof(int));
	hipMalloc(&d_R3,(vertices-1) * sizeof(int));
	hipMalloc(&d_T1, NUMBER_OF_BLOCKS * sizeof(int));
	hipMalloc(&d_T2, NUMBER_OF_BLOCKS * sizeof(int));
	
	// start time
	 hipEvent_t start, stop;
	 float time;
     hipEventCreate(&start);
     hipEventCreate(&stop);
     hipEventRecord( start, 0 );

	// copy host to device graph and initial MST list
		
	hipMemcpy(d_indexArray,h_indexArray, vertices* sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_destinationVertexArray,h_destinationVertexArray, 2 * edges * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_weightArray,h_weightArray,2 * edges * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_nearestVertex,h_nearestVertex,sizeof(int),hipMemcpyHostToDevice);
		
	hipMemcpy(d_R1,h_R1, (vertices - 1)* sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_R2,h_R2, (vertices - 1)* sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_R3,h_R3, (vertices - 1)* sizeof(int),hipMemcpyHostToDevice);


	// MEMSET T1 AND T2
	hipMemset(d_T1,0,NUMBER_OF_BLOCKS * sizeof(int));
	hipMemset(d_T2,0,NUMBER_OF_BLOCKS * sizeof(int));

	if(DEBUG)
	{
		printArray(h_R1,vertices-1);
		printArray(h_R2,vertices-1);
		printArray(h_R3,vertices-1);
	}
	
	
	if( vertices-1 > THREADS_PER_BLOCK)
	{
		secondKernelRequired = 1;
	}
	
	
	

	// kernel launch
	for(int v = 0 ; v < vertices-2; v++)
	{
		findNearestVertexKernelOne<<<NUMBER_OF_BLOCKS,THREADS_PER_BLOCK>>>(d_R2, d_R3, d_T1, d_T2, d_nearestVertex,secondKernelRequired,vertices);

		if(secondKernelRequired)
		{
			if(DEBUG)
			{
				printf("\n Yes Requird \n");
				hipMemcpy(h_T1,d_T1, NUMBER_OF_BLOCKS* sizeof(int),hipMemcpyDeviceToHost);
				hipMemcpy(h_T2,d_T2, NUMBER_OF_BLOCKS * sizeof(int),hipMemcpyDeviceToHost);
				printArray(h_T1,NUMBER_OF_BLOCKS);
				printArray(h_T2,NUMBER_OF_BLOCKS);
			}
			findNearestVertexKernelTwo<<<1,THREADS_PER_BLOCK>>>(d_R1, d_R2, d_R3, d_T1, d_T2, d_nearestVertex);
		}	
		
		if(DEBUG)
		{	
			hipMemcpy(h_R1,d_R1, (vertices - 1)* sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(h_R2,d_R2, (vertices - 1)* sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(h_R3,d_R3, (vertices - 1)* sizeof(int),hipMemcpyDeviceToHost);
			printf("\nAfter Minimum\n");
			printArray(h_R1,vertices-1);
			printArray(h_R2,vertices-1);
			printArray(h_R3,vertices-1);
			printf("%d\n",*h_nearestVertex);
		}
		hipMemcpy(h_nearestVertex,d_nearestVertex,  sizeof(int),hipMemcpyDeviceToHost);
		startIndex = h_indexArray[*h_nearestVertex];
		if(*(h_nearestVertex) == vertices-1)
        		endIndex = 2*edges-1;
	    	else
        		endIndex = h_indexArray[*(h_nearestVertex)+1]-1;
		if(DEBUG)
		{
			printf("\nstart : %d  end : %d\n",startIndex,endIndex);
		}
		
		// update the list
		updateMSTList<<<NUMBER_OF_BLOCKS,THREADS_PER_BLOCK>>>(startIndex, endIndex , d_nearestVertex, d_destinationVertexArray, d_weightArray, d_R1, d_R2, d_R3);
		if(DEBUG)
		{
			hipMemcpy(h_R1,d_R1, (vertices - 1)* sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(h_R2,d_R2, (vertices - 1)* sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(h_R3,d_R3, (vertices - 1)* sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(h_nearestVertex,d_nearestVertex,  sizeof(int),hipMemcpyDeviceToHost);
			printf("\nAfter Update\n");
			printArray(h_R1,vertices-1);
			printArray(h_R2,vertices-1);
			printArray(h_R3,vertices-1);
			printf("%d\n",*h_nearestVertex);
		}
	}

	// for generating the output...
	
	hipMemcpy(h_R1,d_R1, (vertices - 1)* sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(h_R2,d_R2, (vertices - 1)* sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(h_R3,d_R3, (vertices - 1)* sizeof(int),hipMemcpyDeviceToHost);
	
	// end time
	hipEventRecord( stop, 0 );
   	hipEventSynchronize( stop );
   	hipEventElapsedTime( &time, start, stop );
   	hipEventDestroy( start );
   	hipEventDestroy( stop );


	printf("u\tv\tw\n");
	long cost = 0;
	for(int i = 0; i < vertices-1;i++)
	{
		printf("%d\t%d\t%d\t\n",h_R1[i]+1,i+2,h_R3[i]);
		cost = cost + h_R3[i];
	}
	cout<<"\ncost:"<<cost;
		
	printf("\nTime: %fms\n",time);


}
        
